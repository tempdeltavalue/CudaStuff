#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include "common/cpu_anim.h"
#include "common/book.h"

#include "hip/hip_runtime.h"
#include ""

#define DIM 1024
#define PI 3.1415926535897932f
#define MAX_TEMP 1.0f
#define MIN_TEMP 0.0001f
#define SPEED 0.25f



__global__ void copy_const_kernel ( float *iptr, float *cptr, float delta_x) {
	// map from threadIdx/blockIdx to pixel position
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int offset = x + y * blockDim.x * gridDim.x;
	
	if(cptr[offset] != 0) iptr[offset] = cptr[offset];
}
__global__ void blend_kernel(float* outSrc, const float* inSrc, int ticks, float delta_x) {
	// map from threadIdx/blockIdx to pixel position
	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;

	int offset = x + y * blockDim.x * gridDim.x;

	int left = offset - 1;
	int right = offset + 1;
	if (x == 0) left++;
	if (x == DIM - 1) right--;

	int top = offset - DIM;
	int bottom = offset + DIM;
	if (y == 0) top += DIM;
	if (y == DIM - 1) bottom -= DIM;


	float fx = x - DIM / 2; // - delta_x;
	float fy = y - DIM / 2;

	float d = sqrtf(fx * fx + fy * fy);

	unsigned char grey = (unsigned char)(128.0f + 127.0f * cos(d / 10.0f - ticks / 7.0f) / (d / 10.0f + 1.0f));

	outSrc[offset] =  1 * (inSrc[offset] + SPEED * (inSrc[top] + inSrc[bottom] + inSrc[left] + inSrc[right] - inSrc[offset] * 4)) + grey * 0.4;

}


struct DataBlock {
	unsigned char* output_bitmap;
	float* dev_inSrc;
	float* dev_outSrc;
	float* dev_constSrc;

	CPUAnimBitmap* bitmap;
	hipEvent_t start, stop;
	float totalTime;
	float frames;
};

void anim_gpu(DataBlock* d, int ticks) {
	hipEventRecord(d->start, 0);

	dim3 blocks(DIM / 16, DIM / 16);
	dim3 threads(16, 16);

	CPUAnimBitmap* bitmap = d->bitmap;
	printf("b %f", bitmap->deltaX);
	for (int i = 0; i < 200; i++) {
		copy_const_kernel << <blocks, threads >> > (d->dev_inSrc, d->dev_constSrc, 1);

		blend_kernel << <blocks, threads >> > (d->dev_outSrc, d->dev_inSrc, ticks, 1000);

		swap(d->dev_inSrc, d->dev_outSrc);
	}



	float_to_color << <blocks, threads >> > (d->output_bitmap, d->dev_inSrc);

    //recalc squares here
	hipMemcpy(bitmap->get_ptr(), d->output_bitmap, bitmap->image_size(), hipMemcpyDeviceToHost);


	hipEventRecord(d->stop, 0);
	hipEventSynchronize(d->stop);

	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, d->start, d->stop);

	d->totalTime += elapsedTime;
	++d->frames;
}

void anim_exit(DataBlock* d) {
	hipFree(d->dev_inSrc);
	hipFree(d->dev_outSrc);
	hipFree(d->dev_constSrc);

	hipEventDestroy(d->start);
	hipEventDestroy(d->stop);
}

int main() {
	DataBlock data;
	CPUAnimBitmap bitmap(DIM, DIM, &data);
	data.bitmap = &bitmap;
	data.totalTime = 0;
	data.frames = 0;

	hipEventCreate(&data.start);
	hipEventCreate(&data.stop);

	hipMalloc(&data.output_bitmap, bitmap.image_size());

	hipMalloc(&data.dev_inSrc, bitmap.image_size());
	hipMalloc(&data.dev_outSrc, bitmap.image_size());
	hipMalloc(&data.dev_constSrc, bitmap.image_size());

	float* temp = new float[bitmap.image_size()];

	for (int i = 0; i < DIM * DIM; i++) {
		temp[i] = 0;
		int x = i % DIM;
		int y = i / DIM;
		if ((x > 200) && (x < 700) && (y > 210) && (y < 701))
			temp[i] = MAX_TEMP;
	}

	temp[DIM * 100 + 100] = (MAX_TEMP + MIN_TEMP) / 2;
	temp[DIM * 700 + 100] = MIN_TEMP;
	temp[DIM * 300 + 300] = MIN_TEMP;
	temp[DIM * 200 + 700] = MIN_TEMP;

	for (int y = 800; y < 900; y++) {
		for (int x = 400; x < 500; x++) {
			temp[x + y * DIM] = MIN_TEMP;
		}
	}

	hipMemcpy(data.dev_constSrc, temp, bitmap.image_size(), hipMemcpyHostToDevice);


	for (int y = 800; y < DIM; y++) {
		for (int x = 0; x < 200; x++) {
			temp[x + y * DIM] = MAX_TEMP;
		}
	}

	hipMemcpy(data.dev_inSrc, temp, bitmap.image_size(), hipMemcpyHostToDevice);
	free(temp);

	bitmap.anim_and_exit((void(*)(void*, int))anim_gpu, (void(*)(void*)) anim_exit);


	return 0;
}